/*
 * Mandel.cu
 *
 *  Created on: Jan 17, 2016
 *      Author: martin
 *
 *      Trying to setup a window in gtk to show random data produced by the GPU
 */

#include <thrust/device_vector.h>

#include "Application.h"

int main(int argc, char** argv){
	Application app(argc, argv);

	app.start();
	hipDeviceReset();
}


