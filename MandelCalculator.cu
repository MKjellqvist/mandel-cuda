#include "hip/hip_runtime.h"
/*
 * MandelCalculator.cpp
 *
 *  Created on: Jan 17, 2016
 *      Author: martin
 */

#include "MandelCalculator.h"

#include <thrust/device_vector.h>
#include <thrust/complex.h>

MandelCalculator::MandelCalculator(int height, int width):width(width), height(height) {
	int count;
	hipGetDeviceCount(&count);
	if ( !count )
		throw NoDevice();
	hipSetDevice(0);
}

MandelCalculator::~MandelCalculator() {
	hipDeviceSynchronize();
}

typedef thrust::complex<double> complex;

__device__ void abs_squared(const complex& value, double* return_value){
	*return_value = value.imag() * value.imag() + value.real() * value.real();
}
__device__ void iterations_until_divergence(double x, double y, int* result){
	const double DIVERGE_LIMIT = 100.0;
	const int MAX_ITERATIONS = 250;
	complex start(x, y);
	complex current(start);
	*result = 20;
	for(int iterations = 0;iterations < MAX_ITERATIONS; iterations++){
		current *= current;
		current += start;
		double abs;
		abs_squared(current, &abs);
		if( abs > DIVERGE_LIMIT){
			*result = iterations;
			return;
		}
	}
	return;
}
#define N_MANDEL_THREADS 256
__global__ void mandel_calc(color_type* data, int width, int height, double window_height,
		double window_width, double window_top, double window_left){
	int imgIdx  = blockIdx.x * N_MANDEL_THREADS + threadIdx.x;
	int x = imgIdx % width;
	int y = imgIdx / width;

	double dx = window_width * double(x) / width + window_left;
	double dy = window_height * double(y) / height + window_top;
	int iter;
	iterations_until_divergence(dx, dy, &iter);
	data[imgIdx] =  iter * 0x020306;
}

color_vector MandelCalculator::calc(double* view){
	thrust::device_vector<color_type> data(width*height);

	color_type* data_ptr = thrust::raw_pointer_cast(data.data());

	mandel_calc<<<width*height/N_MANDEL_THREADS, N_MANDEL_THREADS>>>(data_ptr, width, height, view[2], view[3], view[0], view[1]);

	color_vector result = data;
	data.clear();
	return result;
}
